
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <hiprand/hiprand.h>

const std::size_t min_num_data = 1u << 10;
const std::size_t max_num_data = 1u << 30;

namespace {
std::string get_rng_name_str(
		const hiprandRngType_t rng_type
		) {
	switch (rng_type) {
#define CASE_RNG_TYPE(rng) case rng: return #rng
		CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_DEFAULT);
		CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_XORWOW);
		CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_MRG32K3A);
		CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_MTGP32);
		CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_MT19937);
		CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
		CASE_RNG_TYPE(HIPRAND_RNG_QUASI_DEFAULT);
		CASE_RNG_TYPE(HIPRAND_RNG_QUASI_SOBOL32);
		CASE_RNG_TYPE(HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL32);
		default:
			return "Unknown";
	}
}
} // noname namespace

void measure_perf(
		const hiprandRngType_t rng_type
		) {
	hiprandGenerator_t curand_gen;
	hiprandCreateGenerator(&curand_gen, rng_type);
	hiprandSetPseudoRandomGeneratorSeed(curand_gen, 0);

	float* data_ptr;
	hipMalloc(&data_ptr, sizeof(float) * max_num_data);

	for (auto num_data = min_num_data; num_data <= max_num_data; num_data <<= 1) {
		hipDeviceSynchronize();
		const auto start_clock = std::chrono::system_clock::now();

		hiprandGenerateUniform(curand_gen, data_ptr, num_data);

		hipDeviceSynchronize();
		const auto end_clock = std::chrono::system_clock::now();

		const auto elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(end_clock - start_clock).count() * 1e-6;

		std::printf("%s,%lu,%e\n",
				get_rng_name_str(rng_type).c_str(),
				num_data,
				elapsed_time
				);
	}
}

int main() {
		measure_perf(HIPRAND_RNG_PSEUDO_DEFAULT);
		measure_perf(HIPRAND_RNG_PSEUDO_XORWOW);
		measure_perf(HIPRAND_RNG_PSEUDO_MRG32K3A);
		measure_perf(HIPRAND_RNG_PSEUDO_MTGP32);
		measure_perf(HIPRAND_RNG_PSEUDO_MT19937);
		measure_perf(HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
		measure_perf(HIPRAND_RNG_QUASI_DEFAULT);
		measure_perf(HIPRAND_RNG_QUASI_SOBOL32);
		measure_perf(HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL32);
}
